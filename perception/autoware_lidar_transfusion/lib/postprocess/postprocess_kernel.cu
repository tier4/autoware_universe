#include "hip/hip_runtime.h"
// Copyright 2024 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "autoware/lidar_transfusion/postprocess/circle_nms_kernel.hpp"
#include "autoware/lidar_transfusion/postprocess/postprocess_kernel.hpp"

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

namespace autoware::lidar_transfusion
{
const size_t THREADS_PER_BLOCK = 256;

struct is_score_greater
{
  is_score_greater(float t) : t_(t) {}

  __device__ bool operator()(const Box3D & b) { return b.score > t_; }

private:
  float t_{0.0};
};

struct is_kept
{
  __device__ bool operator()(const bool keep) { return keep; }
};

struct score_greater
{
  __device__ bool operator()(const Box3D & lb, const Box3D & rb) { return lb.score > rb.score; }
};

__device__ inline float sigmoid(float x)
{
  return 1.0f / (1.0f + expf(-x));
}

__global__ void generateBoxes3D_kernel(
  const float * __restrict__ cls_output, const float * __restrict__ box_output,
  const float * __restrict__ dir_cls_output, const float voxel_size_x, const float voxel_size_y,
  const float min_x_range, const float min_y_range, const int num_proposals, const int num_classes,
  const int num_point_values, const float * __restrict__ yaw_norm_thresholds,
  Box3D * __restrict__ det_boxes3d)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= num_proposals) {
    return;
  }

  int class_id = 0;
  float max_score = cls_output[point_idx];

#pragma unroll
  for (int i = 1; i < num_classes; i++) {
    float score = cls_output[i * num_proposals + point_idx];
    if (score > max_score) {
      max_score = score;
      class_id = i;
    }
  }

  // yaw validation
  const float yaw_sin = dir_cls_output[point_idx];
  const float yaw_cos = dir_cls_output[point_idx + num_proposals];
  const float yaw_norm = sqrtf(yaw_sin * yaw_sin + yaw_cos * yaw_cos);

  det_boxes3d[point_idx].label = class_id;
  det_boxes3d[point_idx].score = yaw_norm >= yaw_norm_thresholds[class_id] ? max_score : 0.f;
  det_boxes3d[point_idx].x = box_output[point_idx] * num_point_values * voxel_size_x + min_x_range;
  det_boxes3d[point_idx].y =
    box_output[point_idx + num_proposals] * num_point_values * voxel_size_y + min_y_range;
  det_boxes3d[point_idx].z = box_output[point_idx + 2 * num_proposals];
  det_boxes3d[point_idx].length = expf(box_output[point_idx + 3 * num_proposals]);
  det_boxes3d[point_idx].width = expf(box_output[point_idx + 4 * num_proposals]);
  det_boxes3d[point_idx].height = expf(box_output[point_idx + 5 * num_proposals]);
  det_boxes3d[point_idx].yaw =
    atan2f(dir_cls_output[point_idx], dir_cls_output[point_idx + num_proposals]);
}

PostprocessCuda::PostprocessCuda(const TransfusionConfig & config, hipStream_t & stream)
: config_(config), stream_(stream)
{
}

// cspell: ignore divup
hipError_t PostprocessCuda::generateDetectedBoxes3D_launch(
  const float * cls_output, const float * box_output, const float * dir_cls_output,
  std::vector<Box3D> & det_boxes3d, hipStream_t stream)
{
  dim3 threads = {THREADS_PER_BLOCK};
  dim3 blocks = {divup(config_.num_proposals_, threads.x)};

  auto boxes3d_d = thrust::device_vector<Box3D>(config_.num_proposals_);
  auto yaw_norm_thresholds_d = thrust::device_vector<float>(
    config_.yaw_norm_thresholds_.begin(), config_.yaw_norm_thresholds_.end());

  generateBoxes3D_kernel<<<blocks, threads, 0, stream>>>(
    cls_output, box_output, dir_cls_output, config_.voxel_x_size_, config_.voxel_y_size_,
    config_.min_x_range_, config_.min_y_range_, config_.num_proposals_, config_.num_classes_,
    config_.num_point_values_, thrust::raw_pointer_cast(yaw_norm_thresholds_d.data()),
    thrust::raw_pointer_cast(boxes3d_d.data()));

  // suppress by score
  const auto num_det_boxes3d = thrust::count_if(
    thrust::device, boxes3d_d.begin(), boxes3d_d.end(), is_score_greater(config_.score_threshold_));
  if (num_det_boxes3d == 0) {
    return hipGetLastError();
  }
  thrust::device_vector<Box3D> det_boxes3d_d(num_det_boxes3d);
  thrust::copy_if(
    thrust::device, boxes3d_d.begin(), boxes3d_d.end(), det_boxes3d_d.begin(),
    is_score_greater(config_.score_threshold_));

  // sort by score
  thrust::sort(det_boxes3d_d.begin(), det_boxes3d_d.end(), score_greater());

  // supress by NMS
  thrust::device_vector<bool> final_keep_mask_d(num_det_boxes3d);
  const auto num_final_det_boxes3d =
    circleNMS(det_boxes3d_d, config_.circle_nms_dist_threshold_, final_keep_mask_d, stream);
  thrust::device_vector<Box3D> final_det_boxes3d_d(num_final_det_boxes3d);
  thrust::copy_if(
    thrust::device, det_boxes3d_d.begin(), det_boxes3d_d.end(), final_keep_mask_d.begin(),
    final_det_boxes3d_d.begin(), is_kept());

  // memcpy device to host
  det_boxes3d.resize(num_final_det_boxes3d);
  thrust::copy(final_det_boxes3d_d.begin(), final_det_boxes3d_d.end(), det_boxes3d.begin());
  return hipGetLastError();
}

}  // namespace autoware::lidar_transfusion
