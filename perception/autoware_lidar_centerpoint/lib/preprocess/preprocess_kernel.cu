#include "hip/hip_runtime.h"
// Copyright 2022 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "autoware/lidar_centerpoint/cuda_utils.hpp"
#include "autoware/lidar_centerpoint/preprocess/preprocess_kernel.hpp"
#include "autoware/lidar_centerpoint/utils.hpp"

#include <cassert>
#include <cmath>

namespace
{
const std::size_t MAX_POINT_IN_VOXEL_SIZE = 32;  // the same as max_point_in_voxel_size_ in config
const std::size_t WARPS_PER_BLOCK = 4;
const std::size_t POINT_DIM_XYZT = 4;
const std::size_t POINT_DIM_XYZIT = 5;
const std::size_t NUM_FEATURES_11 = 11;
}  // namespace

namespace autoware::lidar_centerpoint
{

template <std::size_t POINT_NUM_FEATURES> 
__global__ void generateSweepPoints_kernel(
  const uint8_t * input_points, std::size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, int num_features, float * output_points)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;


  union {
    uint32_t raw{0};
    float value;
  } input_x, input_y, input_z;

  #pragma unroll
  for (int i = 0; i < 4; i++) {  // 4 bytes for float32
    input_x.raw |= input_points[point_idx * input_point_step + i] << i * 8;
    input_y.raw |= input_points[point_idx * input_point_step + i + 4] << i * 8;
    input_z.raw |= input_points[point_idx * input_point_step + i + 8] << i * 8;
  }

  output_points[point_idx * num_features] =
    transform_array[0] * input_x.value + transform_array[4] * input_y.value +
    transform_array[8] * input_z.value + transform_array[12];
  output_points[point_idx * num_features + 1] =
    transform_array[1] * input_x.value + transform_array[5] * input_y.value +
    transform_array[9] * input_z.value + transform_array[13];
  output_points[point_idx * num_features + 2] =
    transform_array[2] * input_x.value + transform_array[6] * input_y.value +
    transform_array[10] * input_z.value + transform_array[14];
  
  if (POINT_NUM_FEATURES == POINT_DIM_XYZT) {
    output_points[point_idx * num_features + 3] = time_lag;
  } else if (POINT_NUM_FEATURES == POINT_DIM_XYZIT) {
    float input_intensity = static_cast<float>(input_points[point_idx * input_point_step + 12]);
    output_points[point_idx * num_features + 3] = input_intensity;
    output_points[point_idx * num_features + 4] = time_lag;
  } else {
    return;
  }
}

hipError_t generateSweepPoints_launch(
  const uint8_t * input_points, std::size_t points_size, int input_point_step, float time_lag,
  const float * transform_array, int num_features, float * output_points, hipStream_t stream)
{
  auto transform_d = cuda::make_unique<float[]>(16);
  CHECK_CUDA_ERROR(hipMemcpyAsync(
    transform_d.get(), transform_array, 16 * sizeof(float), hipMemcpyHostToDevice, stream));

  dim3 blocks((points_size + 256 - 1) / 256);
  dim3 threads(256);
  assert(num_features == POINT_DIM_XYZT || num_features == POINT_DIM_XYZIT);
  
  if (num_features == POINT_DIM_XYZT) {
    generateSweepPoints_kernel<POINT_DIM_XYZT><<<blocks, threads, 0, stream>>>(
      input_points, points_size, input_point_step, time_lag, transform_d.get(), num_features,
      output_points);
  } else if (num_features == POINT_DIM_XYZIT) {
    generateSweepPoints_kernel<POINT_DIM_XYZIT><<<blocks, threads, 0, stream>>>(
      input_points, points_size, input_point_step, time_lag, transform_d.get(), num_features,
      output_points);
  } else {
    throw std::runtime_error("Value of num_features is not supported!");
  }

  hipError_t err = hipGetLastError();
  return err;
}

template <std::size_t POINT_NUM_FEATURES> 
__global__ void shufflePoints_kernel(
  const float * points, const unsigned int * indices, float * shuffled_points,
  const std::size_t points_size, const std::size_t max_size, const std::size_t offset)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= max_size) return;

  int src_idx = indices[(point_idx + offset) % max_size];
  int dst_idx = point_idx;

  if (POINT_NUM_FEATURES == POINT_DIM_XYZT) {
    if (dst_idx >= points_size) {
      shuffled_points[4 * dst_idx + 0] = INFINITY;
      shuffled_points[4 * dst_idx + 1] = INFINITY;
      shuffled_points[4 * dst_idx + 2] = INFINITY;
      shuffled_points[4 * dst_idx + 3] = INFINITY;
    } else {
      shuffled_points[4 * dst_idx + 0] = points[4 * src_idx + 0];
      shuffled_points[4 * dst_idx + 1] = points[4 * src_idx + 1];
      shuffled_points[4 * dst_idx + 2] = points[4 * src_idx + 2];
      shuffled_points[4 * dst_idx + 3] = points[4 * src_idx + 3];
    }
  } else if (POINT_NUM_FEATURES == POINT_DIM_XYZIT) {
    if (dst_idx >= points_size) {
      shuffled_points[5 * dst_idx + 0] = INFINITY;
      shuffled_points[5 * dst_idx + 1] = INFINITY;
      shuffled_points[5 * dst_idx + 2] = INFINITY;
      shuffled_points[5 * dst_idx + 3] = INFINITY;
      shuffled_points[5 * dst_idx + 4] = INFINITY;
    } else {
      shuffled_points[5 * dst_idx + 0] = points[5 * src_idx + 0];
      shuffled_points[5 * dst_idx + 1] = points[5 * src_idx + 1];
      shuffled_points[5 * dst_idx + 2] = points[5 * src_idx + 2];
      shuffled_points[5 * dst_idx + 3] = points[5 * src_idx + 3];
      shuffled_points[5 * dst_idx + 4] = points[5 * src_idx + 4];
    }
  } else {
    return;
  }
}

hipError_t shufflePoints_launch(
  const float * points, const unsigned int * indices, float * shuffled_points,
  const std::size_t points_size, const std::size_t max_size, const std::size_t offset, const int num_features,
  hipStream_t stream)
{
  dim3 blocks((max_size + 256 - 1) / 256);
  dim3 threads(256);

  if (blocks.x == 0) {
    return hipGetLastError();
  }

  if (num_features == POINT_DIM_XYZT) {
    shufflePoints_kernel<POINT_DIM_XYZT><<<blocks, threads, 0, stream>>>(
      points, indices, shuffled_points, points_size, max_size, offset);
  } else if (num_features == POINT_DIM_XYZIT) {
    shufflePoints_kernel<POINT_DIM_XYZIT><<<blocks, threads, 0, stream>>>(
      points, indices, shuffled_points, points_size, max_size, offset);
  } else {
    throw std::runtime_error("Value of num_features is not supported!");
  }
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateIntensityVoxels_random_kernel(
  const float * points, std::size_t points_size, float min_x_range, float max_x_range,
  float min_y_range, float max_y_range, float min_z_range, float max_z_range, float pillar_x_size,
  float pillar_y_size, float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask,
  float * voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  float x = points[point_idx * 5];
  float y = points[point_idx * 5 + 1];
  float z = points[point_idx * 5 + 2];
  float i = points[point_idx * 5 + 3];
  float t = points[point_idx * 5 + 4];

  if (
    x < min_x_range || x >= max_x_range || y < min_y_range || y >= max_y_range ||
    z < min_z_range || z >= max_z_range)
    return;

  int voxel_idx = floorf((x - min_x_range) / pillar_x_size);
  int voxel_idy = floorf((y - min_y_range) / pillar_y_size);
  voxel_idx = voxel_idx < 0 ? 0 : voxel_idx >= grid_x_size ? grid_x_size - 1 : voxel_idx;
  voxel_idy = voxel_idy < 0 ? 0 : voxel_idy >= grid_y_size ? grid_y_size - 1 : voxel_idy;
  unsigned int voxel_index = (grid_x_size - 1 - voxel_idx) * grid_y_size + voxel_idy;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if (point_id >= MAX_POINT_IN_VOXEL_SIZE) return;
  float * address = voxels + (voxel_index * MAX_POINT_IN_VOXEL_SIZE + point_id) * POINT_DIM_XYZIT;
  atomicExch(address + 0, x);
  atomicExch(address + 1, y);
  atomicExch(address + 2, z);
  atomicExch(address + 3, i);
  atomicExch(address + 4, t);
}

__global__ void generateVoxels_random_kernel(
  const float * points, std::size_t points_size, float min_x_range, float max_x_range,
  float min_y_range, float max_y_range, float min_z_range, float max_z_range, float pillar_x_size,
  float pillar_y_size, float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask,
  float * voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  float4 point = ((float4 *)points)[point_idx];

  if (
    point.x < min_x_range || point.x >= max_x_range || point.y < min_y_range ||
    point.y >= max_y_range || point.z < min_z_range || point.z >= max_z_range)
    return;

  int voxel_idx = floorf((point.x - min_x_range) / pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range) / pillar_y_size);
  voxel_idx = voxel_idx < 0 ? 0 : voxel_idx >= grid_x_size ? grid_x_size - 1 : voxel_idx;
  voxel_idy = voxel_idy < 0 ? 0 : voxel_idy >= grid_y_size ? grid_y_size - 1 : voxel_idy;
  unsigned int voxel_index = (grid_x_size - 1 - voxel_idx) * grid_y_size + voxel_idy;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if (point_id >= MAX_POINT_IN_VOXEL_SIZE) return;
  float * address = voxels + (voxel_index * MAX_POINT_IN_VOXEL_SIZE + point_id) * 4;
  atomicExch(address + 0, point.x);
  atomicExch(address + 1, point.y);
  atomicExch(address + 2, point.z);
  atomicExch(address + 3, point.w);
}

hipError_t generateVoxels_random_launch(
  const float * points, std::size_t points_size, float min_x_range, float max_x_range,
  float min_y_range, float max_y_range, float min_z_range, float max_z_range, float pillar_x_size,
  float pillar_y_size, float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask,
  float * voxels, const int num_features, hipStream_t stream)
{
  dim3 blocks((points_size + 256 - 1) / 256);
  dim3 threads(256);

  if (blocks.x == 0) {
    return hipGetLastError();
  }

  if (num_features == POINT_DIM_XYZT) {
    generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>(
      points, points_size, min_x_range, max_x_range, min_y_range, max_y_range, min_z_range,
      max_z_range, pillar_x_size, pillar_y_size, pillar_z_size, grid_y_size, grid_x_size, mask,
      voxels);
  } else if (num_features == POINT_DIM_XYZIT) {
    generateIntensityVoxels_random_kernel<<<blocks, threads, 0, stream>>>(
      points, points_size, min_x_range, max_x_range, min_y_range, max_y_range, min_z_range,
      max_z_range, pillar_x_size, pillar_y_size, pillar_z_size, grid_y_size, grid_x_size, mask,
      voxels);
  } else {
    throw std::runtime_error("Value of num_features is not supported!");
  }
  hipError_t err = hipGetLastError();
  return err;
}

template <std::size_t POINT_NUM_FEATURES>
__global__ void generateBaseFeatures_kernel(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs)
{
  // exchange x and y to process in a row-major order
  // flip x axis direction to process front to back
  unsigned int voxel_idx_inverted = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int voxel_idy = blockIdx.x * blockDim.x + threadIdx.x;
  if (voxel_idx_inverted >= grid_x_size || voxel_idy >= grid_y_size) return;
  unsigned int voxel_idx = grid_x_size - 1 - voxel_idx_inverted;

  unsigned int voxel_index = voxel_idx_inverted * grid_y_size + voxel_idy;
  unsigned int count = mask[voxel_index];
  if (!(count > 0)) return;
  count = count < MAX_POINT_IN_VOXEL_SIZE ? count : MAX_POINT_IN_VOXEL_SIZE;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);
  if (current_pillarId > max_voxel_size - 1) return;

  voxel_num[current_pillarId] = count;

  uint3 idx = {0, voxel_idy, voxel_idx};
  ((uint3 *)voxel_idxs)[current_pillarId] = idx;

  for (int i = 0; i < count; i++) {
    int inIndex = voxel_index * MAX_POINT_IN_VOXEL_SIZE + i;
    int outIndex = current_pillarId * MAX_POINT_IN_VOXEL_SIZE + i;
    if (POINT_NUM_FEATURES == POINT_DIM_XYZT) {
      ((float4 *)voxel_features)[outIndex] = ((float4 *)voxels)[inIndex];
    } else if (POINT_NUM_FEATURES == POINT_DIM_XYZIT) {
      voxel_features[outIndex * 5] = voxels[inIndex * 5];
      voxel_features[outIndex * 5 + 1] = voxels[inIndex * 5 + 1];
      voxel_features[outIndex * 5 + 2] = voxels[inIndex * 5 + 2];
      voxel_features[outIndex * 5 + 3] = voxels[inIndex * 5 + 3];
      voxel_features[outIndex * 5 + 4] = voxels[inIndex * 5 + 4];
    }
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t generateBaseFeatures_launch(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs, const int num_features,
  hipStream_t stream)
{
  // exchange x and y to process in a row-major order
  dim3 threads = {32, 32};
  dim3 blocks = {
    (grid_y_size + threads.x - 1) / threads.x, (grid_x_size + threads.y - 1) / threads.y};
  
  if (num_features == POINT_DIM_XYZT) {
    generateBaseFeatures_kernel<POINT_DIM_XYZT><<<blocks, threads, 0, stream>>>(
      mask, voxels, grid_y_size, grid_x_size, max_voxel_size, pillar_num, voxel_features, voxel_num,
      voxel_idxs);
  } else if (num_features  == POINT_DIM_XYZIT) {
    generateBaseFeatures_kernel<POINT_DIM_XYZIT><<<blocks, threads, 0, stream>>>(
      mask, voxels, grid_y_size, grid_x_size, max_voxel_size, pillar_num, voxel_features, voxel_num,
      voxel_idxs);
  } else {
    throw std::runtime_error("Value of num_features is not supported!");
  }
  hipError_t err = hipGetLastError();
  return err;
}

template <std::size_t ENCODER_IN_FEATURE_SIZE>
__global__ void generateFeatures_kernel(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const float voxel_x, const float voxel_y, const float voxel_z,
  const float range_min_x, const float range_min_y, const float range_min_z, float * features)
{
  // voxel_features (float): (max_voxel_size, max_point_in_voxel_size, point_feature_size)
  // voxel_num_points (int): (max_voxel_size)
  // coords (int): (max_voxel_size, point_dim_size)
  int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;
  int point_idx = threadIdx.x % MAX_POINT_IN_VOXEL_SIZE;
  int pillar_idx_inBlock = threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;  // max_point_in_voxel_size

  unsigned int num_pillars = num_voxels[0];
  if (pillar_idx >= num_pillars) return;

  // load src
  __shared__ float4 pillarSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE];
  __shared__ float3 pillarSumSM[WARPS_PER_BLOCK];
  __shared__ int3 cordsSM[WARPS_PER_BLOCK];
  __shared__ int pointsNumSM[WARPS_PER_BLOCK];
  __shared__ float pillarOutSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE][ENCODER_IN_FEATURE_SIZE];

  if (threadIdx.x < WARPS_PER_BLOCK) {
    pointsNumSM[threadIdx.x] = voxel_num_points[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    cordsSM[threadIdx.x] = ((int3 *)coords)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    pillarSumSM[threadIdx.x] = {0, 0, 0};
  }

  pillarSM[pillar_idx_inBlock][point_idx] =
    ((float4 *)voxel_features)[pillar_idx * MAX_POINT_IN_VOXEL_SIZE + point_idx];
  __syncthreads();

  // calculate sm in a pillar
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x), pillarSM[pillar_idx_inBlock][point_idx].x);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y), pillarSM[pillar_idx_inBlock][point_idx].y);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z), pillarSM[pillar_idx_inBlock][point_idx].z);
  }
  __syncthreads();

  // feature-mean
  float3 mean;
  float validPoints = pointsNumSM[pillar_idx_inBlock];
  mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
  mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
  mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

  mean.x = pillarSM[pillar_idx_inBlock][point_idx].x - mean.x;
  mean.y = pillarSM[pillar_idx_inBlock][point_idx].y - mean.y;
  mean.z = pillarSM[pillar_idx_inBlock][point_idx].z - mean.z;

  // calculate offset
  float x_offset = voxel_x / 2 + cordsSM[pillar_idx_inBlock].z * voxel_x + range_min_x;
  float y_offset = voxel_y / 2 + cordsSM[pillar_idx_inBlock].y * voxel_y + range_min_y;
  float z_offset = voxel_z / 2 + cordsSM[pillar_idx_inBlock].x * voxel_z + range_min_z;

  // feature-offset
  float3 center;
  center.x = pillarSM[pillar_idx_inBlock][point_idx].x - x_offset;
  center.y = pillarSM[pillar_idx_inBlock][point_idx].y - y_offset;
  center.z = pillarSM[pillar_idx_inBlock][point_idx].z - z_offset;

  // store output
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = pillarSM[pillar_idx_inBlock][point_idx].x;
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = pillarSM[pillar_idx_inBlock][point_idx].y;
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = pillarSM[pillar_idx_inBlock][point_idx].z;
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = pillarSM[pillar_idx_inBlock][point_idx].w;

    pillarOutSM[pillar_idx_inBlock][point_idx][4] = mean.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][5] = mean.y;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = mean.z;

    pillarOutSM[pillar_idx_inBlock][point_idx][7] = center.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][8] = center.y;

    if (ENCODER_IN_FEATURE_SIZE == 10) {
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = center.z;
    }

  } else {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;

    pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;

    pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;

    if (ENCODER_IN_FEATURE_SIZE == 10) {
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = 0;
    }
  }

  __syncthreads();

  for (int i = 0; i < ENCODER_IN_FEATURE_SIZE; i++) {
    int outputSMId = pillar_idx_inBlock * MAX_POINT_IN_VOXEL_SIZE * ENCODER_IN_FEATURE_SIZE +
                     i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    int outputId = pillar_idx * MAX_POINT_IN_VOXEL_SIZE * ENCODER_IN_FEATURE_SIZE +
                   i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    features[outputId] = ((float *)pillarOutSM)[outputSMId];
  }
}

__global__ void generateIntensityFeatures_kernel(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const float voxel_x, const float voxel_y, const float voxel_z,
  const float range_min_x, const float range_min_y, const float range_min_z, float * features)
{
  // voxel_features (float): (max_voxel_size, max_point_in_voxel_size, point_feature_size)
  // voxel_num_points (int): (max_voxel_size)
  // coords (int): (max_voxel_size, point_dim_size)
  int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;
  int point_idx = threadIdx.x % MAX_POINT_IN_VOXEL_SIZE;
  int pillar_idx_inBlock = threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;  // max_point_in_voxel_size

  unsigned int num_pillars = num_voxels[0];
  if (pillar_idx >= num_pillars) return;

  // load src
  __shared__ float pillarSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE][POINT_DIM_XYZIT];
  __shared__ float3 pillarSumSM[WARPS_PER_BLOCK];
  __shared__ int3 cordsSM[WARPS_PER_BLOCK];
  __shared__ int pointsNumSM[WARPS_PER_BLOCK];
  __shared__ float pillarOutSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE][NUM_FEATURES_11];

  if (threadIdx.x < WARPS_PER_BLOCK) {
    pointsNumSM[threadIdx.x] = voxel_num_points[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    cordsSM[threadIdx.x] = ((int3 *)coords)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    pillarSumSM[threadIdx.x] = {0, 0, 0};
  }

//   pillarSM[pillar_idx_inBlock][point_idx][0] = voxel_features[pillar_idx * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + point_idx * POINT_DIM_XYZIT];
//   pillarSM[pillar_idx_inBlock][point_idx][1] = voxel_features[pillar_idx * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + point_idx * POINT_DIM_XYZIT + 1];
//   pillarSM[pillar_idx_inBlock][point_idx][2] = voxel_features[pillar_idx * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + point_idx * POINT_DIM_XYZIT + 2];
//   pillarSM[pillar_idx_inBlock][point_idx][3] = voxel_features[pillar_idx * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + point_idx * POINT_DIM_XYZIT + 3];
//   pillarSM[pillar_idx_inBlock][point_idx][4] = voxel_features[pillar_idx * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + point_idx * POINT_DIM_XYZIT + 4];
  
  #pragma unroll
  for (int i = 0; i < POINT_DIM_XYZIT; i++) {  
	int pillarSMId = pillar_idx_inBlock * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    int voxel_feature_id = pillar_idx * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
	((float *)pillarSM)[pillarSMId] = ((float *)voxel_features)[voxel_feature_id];
  	// pillarSM[pillar_idx_inBlock][point_idx][0] = voxel_features[pillar_idx * MAX_POINT_IN_VOXEL_SIZE * POINT_DIM_XYZIT + point_idx * POINT_DIM_XYZIT];
  }
  __syncthreads();

  // calculate sm in a pillar
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x), pillarSM[pillar_idx_inBlock][point_idx][0]);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y), pillarSM[pillar_idx_inBlock][point_idx][1]);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z), pillarSM[pillar_idx_inBlock][point_idx][2]);
  }
  __syncthreads();

  // feature-mean
  float3 mean;
  float validPoints = pointsNumSM[pillar_idx_inBlock];
  mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
  mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
  mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

  mean.x = pillarSM[pillar_idx_inBlock][point_idx][0] - mean.x;
  mean.y = pillarSM[pillar_idx_inBlock][point_idx][1] - mean.y;
  mean.z = pillarSM[pillar_idx_inBlock][point_idx][2] - mean.z;

  // calculate offset
  float x_offset = voxel_x / 2 + cordsSM[pillar_idx_inBlock].z * voxel_x + range_min_x;
  float y_offset = voxel_y / 2 + cordsSM[pillar_idx_inBlock].y * voxel_y + range_min_y;
  float z_offset = voxel_z / 2 + cordsSM[pillar_idx_inBlock].x * voxel_z + range_min_z;

  // feature-offset
  float3 center;
  center.x = pillarSM[pillar_idx_inBlock][point_idx][0] - x_offset;
  center.y = pillarSM[pillar_idx_inBlock][point_idx][1] - y_offset;
  center.z = pillarSM[pillar_idx_inBlock][point_idx][2] - z_offset;

  // store output
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = pillarSM[pillar_idx_inBlock][point_idx][0];
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = pillarSM[pillar_idx_inBlock][point_idx][1];
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = pillarSM[pillar_idx_inBlock][point_idx][2];
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = pillarSM[pillar_idx_inBlock][point_idx][3];
    pillarOutSM[pillar_idx_inBlock][point_idx][4] = pillarSM[pillar_idx_inBlock][point_idx][4];

    pillarOutSM[pillar_idx_inBlock][point_idx][5] = mean.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = mean.y;
    pillarOutSM[pillar_idx_inBlock][point_idx][7] = mean.z;

    pillarOutSM[pillar_idx_inBlock][point_idx][8] = center.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][9] = center.y;
    pillarOutSM[pillar_idx_inBlock][point_idx][10] = center.z;
  } else {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
    
    pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;

    pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][9] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][10] = 0;
  }

  __syncthreads();

  for (int i = 0; i < NUM_FEATURES_11; i++) {
    int outputSMId = pillar_idx_inBlock * MAX_POINT_IN_VOXEL_SIZE * NUM_FEATURES_11 +
                     i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    int outputId = pillar_idx * MAX_POINT_IN_VOXEL_SIZE * NUM_FEATURES_11 +
                   i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    features[outputId] = ((float *)pillarOutSM)[outputSMId];
  }
}

// cspell: ignore divup
hipError_t generateFeatures_launch(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const std::size_t max_voxel_size, const float voxel_size_x,
  const float voxel_size_y, const float voxel_size_z, const float range_min_x,
  const float range_min_y, const float range_min_z, float * features, 
  const std::size_t encoder_in_feature_size, hipStream_t stream)
{
  dim3 blocks(divup(max_voxel_size, WARPS_PER_BLOCK));
  dim3 threads(WARPS_PER_BLOCK * MAX_POINT_IN_VOXEL_SIZE);

  // No intensity and no distance of point cloud to voxel_z 
  if (encoder_in_feature_size == 9) {
    generateFeatures_kernel<9><<<blocks, threads, 0, stream>>>(
      voxel_features, voxel_num_points, coords, num_voxels, voxel_size_x, voxel_size_y, voxel_size_z,
      range_min_x, range_min_y, range_min_z, features);
  // No intensity, but include distance of point cloud to voxel_z 
  } else if (encoder_in_feature_size == 10) {
    generateFeatures_kernel<10><<<blocks, threads, 0, stream>>>(
      voxel_features, voxel_num_points, coords, num_voxels, voxel_size_x, voxel_size_y, voxel_size_z,
      range_min_x, range_min_y, range_min_z, features);
  // Intensity, and include distance of point cloud to voxel_z 
  } else if (encoder_in_feature_size == NUM_FEATURES_11) {
    generateIntensityFeatures_kernel<<<blocks, threads, 0, stream>>>(
      voxel_features, voxel_num_points, coords, num_voxels, voxel_size_x, voxel_size_y, voxel_size_z,
      range_min_x, range_min_y, range_min_z, features);
  } else {
    throw std::runtime_error("Value of encoder_in_feature_size is not supported!");
  }
  return hipGetLastError();
}

}  // namespace autoware::lidar_centerpoint
